#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdlib.h>
#include <random>

#include "../../codes/common/cpu_anim.h"
#include "../../codes/common/book.h"


//This code is highly inspired by the codes found in 
//chapter 8 from "CUDA by example - an introduction to general purpose GPU programming"
//written by Jason Sanders and Edward Kandrot.
//The necessary headers, cpu_anim.h and book.h are used in the book and therefore used here as well.

texture<float, 2>  texIn;

struct DataBlock //To store necessary data
{
	int dim;
	double temp;
	unsigned char   *output_bitmap;
	float           *dev_inSrc;
	CPUAnimBitmap  *bitmap;

};

__global__ void kernel_spins(float *dst, int rand_x, int rand_y, double rand_energy,double temp) 
{
	//Performing a MC-cycle at one thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x == rand_x && y == rand_y)
	{
		double  spin_top, spin_left, spin_this, spin_right, spin_bottom;
		double w[17];
		for (int i = 0; i < 17; i++) w[i] = 0;
		for (int i = -8; i < 9; i += 4) w[i + 8] = exp(-((double)i) / temp);

		
		spin_top = tex2D(texIn, x, y - 1);
		spin_left = tex2D(texIn, x - 1, y);
		spin_this = tex2D(texIn, x, y);
		spin_right = tex2D(texIn, x + 1, y);
		spin_bottom = tex2D(texIn, x, y + 1);
		

		int deltaEnergy = 2 * spin_this*
			(spin_left +
			spin_right +
			spin_top +
			spin_bottom);
		if (rand_energy <= w[deltaEnergy+8])
		{
			int offset = x + y * blockDim.x * gridDim.x;
			dst[offset] = spin_this == 1 ? 0 : 1;

		}
		
	}
	
} //End: kernel_spins


void anim_gpu(DataBlock *d,int ticks) 
{
	int numSpins = d->dim;
	dim3    blocks(numSpins / 16, numSpins / 16);
	dim3    threads(16, 16);
	CPUAnimBitmap  *bitmap = d->bitmap;

	std::random_device rd;
	std::mt19937_64 gen(rd());
	std::uniform_real_distribution<double> distr(0.0, 1.0);

	int cycle = 2000 + ticks * 2000;
	double temp = d->temp;
	for (int i = 0; i < 2000; i++)
	{
		float * out = d->dev_inSrc;
		int rand_x = (int)(distr(gen)*(double)numSpins);
		int rand_y = (int)(distr(gen)*(double)numSpins);
		double rand_energy = distr(gen);
		kernel_spins << <blocks, threads >> >(out, rand_x, rand_y, rand_energy, temp);
	}
	
	float_to_color << <blocks, threads >> >(d->output_bitmap,
		d->dev_inSrc);

	//Render the results after the MC-cycles
	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(),
		d->output_bitmap,
		bitmap->image_size(),
		hipMemcpyDeviceToHost));

	std::cout << "Cycle: " << cycle << std::endl;

} //End: anim_gpu

void anim_exit(DataBlock *d) 
{
	hipUnbindTexture(texIn);
	HANDLE_ERROR(hipFree(d->dev_inSrc));
} //End: anim_exit


int main(void) 
{
	DataBlock   data;

	int numSpins;
	std::cout << "Number of spins in each dimension: ";
	std::cin >> numSpins;
	std::cout << "Temperature: ";
	double temp;
	std::cin >> temp;

	CPUAnimBitmap bitmap(numSpins, numSpins, &data);
	data.temp = temp;
	data.dim = numSpins;
	data.bitmap = &bitmap;

	int imageSize = bitmap.image_size();

	HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap,
		imageSize));

	HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc,
		imageSize));

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	HANDLE_ERROR(hipBindTexture2D(NULL, texIn,
		data.dev_inSrc,
		desc, numSpins, numSpins,
		sizeof(float) * numSpins));


	//Initilizing at a ground state
	float *spins = (float*)malloc(imageSize);

	for (int i = 0; i<numSpins*numSpins; i++)
	{
		spins[i] = 1;
	}

	//Send over the ground state to the GPU
	HANDLE_ERROR(hipMemcpy(data.dev_inSrc, spins,
		imageSize,
		hipMemcpyHostToDevice));
	free(spins);

	//Render and show
	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu,
		(void(*)(void*))anim_exit);
	return 0;

}//End: main